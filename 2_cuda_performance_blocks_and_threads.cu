#include "hip/hip_runtime.h"
#include <stdio.h>
const int arraySize=20;
const int Maxblock=10;
const int Maxthread=10;
__global__ void kernelfunction(int*a,int*b,int*c){
    int tid = threadIdx.x+blockIdx.x*blockDim.x;//计算线程索引
	while(tid < arraySize){
		c[tid] = a[tid]+b[tid];
		tid += gridDim.x*blockDim.x;
	}
}

int main(void){
    printf("Cuda_Performance block threads\n");

    int a[arraySize],b[arraySize],c[arraySize];
    int *d_a,*d_b,*d_c;
    int size =sizeof(int);
// Give variables values
    int i = 0;
    for(i=0;i<arraySize;i++){  
        a[i] = i;
        b[i] = i;
    }  
// Create Status for error check
    hipError_t cudastatus;
    hipDeviceProp_t prop;  
// Get Cuda Numbers
// 获取cuda数目  
    int count;  
    hipGetDeviceCount(&count);  
    printf("CUDA Numbers==%d\n",count);  
    for(i=0;i<count;i++){  
    // Get device properties
    // 获取设备的属性  
        hipGetDeviceProperties(&prop,i);  
        printf("compute capability: %d.%d\n",prop.major,prop.minor);  
        printf("Shared mem per mp: %ld\n",prop.sharedMemPerBlock);  
        printf("warp size: %d.%d\n",prop.warpSize);  
        printf("maxThreadsPerMultiProcessor: %ld\n",prop.maxThreadsPerMultiProcessor;) 
    }  

// loop for testing different block & threads
    int block,thread;
	for(block = 0;block<Maxblock;block++){
	for(thread = 0;thread<Maxthread;thread++){

// Allocate space for device
// take the address of d_a,and cast into void**
// 取d_a的地址（一个二级指针），然后类型转换成void**
    cudastatus=hipMalloc((void **)&d_a, arraySize*size);
    cudastatus=hipMalloc((void **)&d_b, arraySize*size);
    cudastatus=hipMalloc((void **)&d_c, arraySize*size);

// Timing Start 
    hipEvent_t start, stop;
    float timeall;
    cudastatus=hipEventCreate(&start);
    cudastatus=hipEventCreate(&stop);
    cudastatus=hipEventRecord( start, 0 );
// CopyToGPU
    cudastatus=hipMemcpy(d_a,a,arraySize*size,hipMemcpyHostToDevice);
    cudastatus=hipMemcpy(d_b,b,arraySize*size,hipMemcpyHostToDevice);
    kernelfunction<<<block,thread>>>(d_a,d_b,d_c);
    cudastatus=hipMemcpy(&c,d_c,arraySize*size,hipMemcpyDeviceToHost);
// Timing End
    cudastatus=hipEventRecord( stop, 0 );
    cudastatus=hipEventSynchronize( stop );
    cudastatus=hipEventElapsedTime( &timeall, start, stop );
    cudastatus=hipEventDestroy( start );
    cudastatus=hipEventDestroy( stop );
/*
	for(i = 0;i<arraySize;i++)
		printf("%i+%i=%i\n",a[i],b[i],c[i]);
*/
    printf("block: %i thread %i  time:%f \n",block,thread,timeall);
    cudastatus=hipFree(d_a);
    cudastatus=hipFree(d_b);
    cudastatus=hipFree(d_c);
    if (cudastatus != hipSuccess) {
         fprintf(stderr, "Failed at loop block %i thread %i %s\n",block,thread,hipGetErrorString(cudastatus));
    }
    }
    }
    if (cudastatus != hipSuccess) {
         fprintf(stderr, "Failed %s\n", hipGetErrorString(cudastatus));
    }
    return 0;
}
