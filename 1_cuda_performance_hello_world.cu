
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernelfunction(int*a,int*b,int*c){
    *c=*a+*b;
}

int main(void){
    printf("Cuda_Performance Hello World\n");
    int a,b,c;
    int *d_a,*d_b,*d_c;
    int size =sizeof(int);
// take the address of d_a,and cast into void**
// 取d_a的地址（一个二级指针），然后类型转换成void**
// Allocate space for device

// Create Status for error check
    hipError_t cudastatus;

    cudastatus=hipMalloc((void **)&d_a, size);
    cudastatus=hipMalloc((void **)&d_b, size);
    cudastatus=hipMalloc((void **)&d_c, size);
    
    a = 1;
    b = 2;

    hipEvent_t start, stop;
    float timeall;
    cudastatus=hipEventCreate(&start);
    cudastatus=hipEventCreate(&stop);
    cudastatus=hipEventRecord( start, 0 );

    cudastatus=hipMemcpy(d_a,&a,size,hipMemcpyHostToDevice);
    cudastatus=hipMemcpy(d_b,&b,size,hipMemcpyHostToDevice);
    kernelfunction<<<1,1>>>(d_a,d_b,d_c);
    cudastatus=hipMemcpy(&c,d_c,size,hipMemcpyDeviceToHost);

    cudastatus=hipEventRecord( stop, 0 );
    cudastatus=hipEventSynchronize( stop );
    cudastatus=hipEventElapsedTime( &timeall, start, stop );
    cudastatus=hipEventDestroy( start );
    cudastatus=hipEventDestroy( stop );

    printf("c:%i \n",c);
    printf("time:%f \n",timeall);
    cudastatus=hipFree(d_a);
    cudastatus=hipFree(d_b);
    cudastatus=hipFree(d_c);

    if (cudastatus != hipSuccess) {
         fprintf(stderr, "Failed %s\n", hipGetErrorString(cudastatus));
    }
    return 0;
}
