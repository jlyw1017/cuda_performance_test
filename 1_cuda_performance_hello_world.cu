
#include <hip/hip_runtime.h>
#include <stdio.h>

// 这个是kernel函数，就是GPU函数
__global__ void kernelfunction(int*a,int*b,int*c){
    *c=*a+*b;
}

int main(void){
    printf("Cuda_Performance Hello World\n");
    int a,b,c;
    int *d_a,*d_b,*d_c;
    int size =sizeof(int);
// take the address of d_a,and cast into void**
// 取d_a的地址（一个二级指针），然后类型转换成void**

// Create Status for error check
// 这个是为了错误检查
    hipError_t cudastatus;
// Allocate space for device
// 分配gpu内存
    cudastatus=hipMalloc((void **)&d_a, size);
    cudastatus=hipMalloc((void **)&d_b, size);
    cudastatus=hipMalloc((void **)&d_c, size);
    
    a = 1;
    b = 2;
// Start Timing
// 计时模块
    hipEvent_t start, stop;
    float timeall;
    cudastatus=hipEventCreate(&start);
    cudastatus=hipEventCreate(&stop);
    cudastatus=hipEventRecord( start, 0 );
// CopyToGPU
// 上传到GPU
    cudastatus=hipMemcpy(d_a,&a,size,hipMemcpyHostToDevice);
    cudastatus=hipMemcpy(d_b,&b,size,hipMemcpyHostToDevice);
    kernelfunction<<<1,1>>>(d_a,d_b,d_c);
    cudastatus=hipMemcpy(&c,d_c,size,hipMemcpyDeviceToHost);

// Timing
// 计时结束
    cudastatus=hipEventRecord( stop, 0 );
    cudastatus=hipEventSynchronize( stop );
    cudastatus=hipEventElapsedTime( &timeall, start, stop );
    cudastatus=hipEventDestroy( start );
    cudastatus=hipEventDestroy( stop );

    printf("c:%i \n",c);
    printf("time:%f \n",timeall);
   
 // 释放内存
    cudastatus=hipFree(d_a);
    cudastatus=hipFree(d_b);
    cudastatus=hipFree(d_c);

    if (cudastatus != hipSuccess) {
         fprintf(stderr, "Failed %s\n", hipGetErrorString(cudastatus));
    }
    return 0;
}
